#include "hip/hip_runtime.h"
#include "ScreenHelperCUDA.h"
#include "BOT.h"
#include "Params.h"

#include <stdio.h>

using namespace std;

__constant__ float MATCH_THRESHOLD;

__device__ __forceinline__ int matches(int num, int area) {
    return (float)num / area >= MATCH_THRESHOLD;
};

__device__ __forceinline__ int posToIdx(int x, int y, dim3 tabSize) {
    return y * tabSize.x + x;
}

__device__ float3 colorAt(float* tab, int x, int y, dim3 tabSize) {
    int idx = posToIdx(x, y, tabSize);
    float3 ans;
    ans.x = tab[3 * idx];
    ans.x = tab[3 * idx + 1];
    ans.x = tab[3 * idx + 2];
    return ans;
}

__device__ __forceinline__ float3 colorAt(float3* tab, int x, int y, dim3 tabSize) {
    return tab[posToIdx(x, y, tabSize)];
}

__global__ void matchRect(float* screen, dim3 screenSize, float* rect, dim3 rectSize, int* ans) {
    int idx = blockIdx.x * blockDim.x  + threadIdx.x;
    int hit = 0, cnt = 0;
    int startPointY = idx / screenSize.x;
    int startPointX = idx % screenSize.x;
    int rectArea = rectSize.x * rectSize.y;
    if (idx >= screenSize.x * screenSize.y) {
        return;
    }

    for (int y = startPointY; y < startPointY + rectSize.y && y < screenSize.y; y++) {
        for (int x = startPointX; x < startPointX + rectSize.x && x < screenSize.x; x++) {
            float3 sc = colorAt(screen, x, y, screenSize);
            float3 rc = colorAt(rect, x - startPointX, y - startPointY, rectSize);
            
            hit += sc.x == rc.x && sc.y == rc.y && sc.z == rc.z;
            cnt++;

            int possibleBest = hit + rectArea - cnt;
            if (!matches(possibleBest, rectArea)) {
                ans[idx] = 0;
                return;
            }
        }
    }
    
    ans[idx] = matches(hit, rectArea);
}

__global__ void matchRect32(float* globalScreen, dim3 globalScreenSize, float* rect, dim3 rectSize, int* ans) {
    int threadStartY = threadIdx.x / 32;
    int threadStartX = threadIdx.x % 32;
    dim3 screenSize = dim3(32 * 2, 32 * 2);
    __shared__ float3 screen[32 * 2 * 32 * 2];
    int rectArea = rectSize.x * rectSize.y;
    
    int blockLength = 32;
    int blocksPerLine = ceilf(globalScreenSize.x / blockLength);
    int blockStartY = (blockIdx.x / blocksPerLine) * blockLength;
    int blockStartX = (blockIdx.x % blocksPerLine) * blockLength;
    if (blockStartX >= globalScreenSize.x || blockStartY >= globalScreenSize.y) {
        return;
    }

    for (int y = threadStartY; y < screenSize.y; y += 32) {
        for (int x = threadStartX; x < screenSize.x; x += 32) {
            int globalX = blockStartX + x;
            int globalY = blockStartY + y;
            if (globalX < globalScreenSize.x && globalY < globalScreenSize.y) {
                screen[posToIdx(x, y, screenSize)] = colorAt(globalScreen, globalX, globalY, globalScreenSize);
            }
        }
    }
    __syncthreads();

    int hit = 0;
    int cnt = 0;
    int idx = posToIdx(blockStartX + threadStartX, blockStartY + threadStartY, globalScreenSize);
    for (int y = threadStartY; y < threadStartY + rectSize.y; y++) {
        for (int x = threadStartX; x < threadStartX + rectSize.x; x++) {
            float3 sc = colorAt(screen, x, y, screenSize);
            float3 rc = colorAt(rect, x - threadStartX, y - threadStartY, rectSize);

            hit += sc.x == rc.x && sc.y == rc.y && sc.z == rc.z;
            cnt++;

            int possibleBest = hit + rectArea - cnt;
            if (!matches(possibleBest, rectArea)) {
                ans[idx] = 0;
                return;
            }
        }
    }

    ans[idx] = matches(hit, rectArea);
}

constexpr int MAX_ANS_SIZE = 1920 * 1080 * 42;
int ans[MAX_ANS_SIZE];

Point ScreenHelperCUDA::find(const BmpRect& rect, bool forceSlow) {
    const Screen& screen = BOT::screen;

    // Choose which GPU to run on, change this on a multi-GPU system.
    if (hipSetDevice(0) != hipSuccess) {
        Logger::error("ScreenHelperCUDA::find()", "hipSetDevice failed!");
    }

    float* h_screen = reinterpret_cast<float*>(screen._screen);
    float* h_rect = reinterpret_cast<float*>(rect._colors);

    dim3 screenSize = dim3(screen.width(), screen.height());
    dim3 rectSize = dim3(rect.width(), rect.height());
    int ansSize = screen.size();

    float* d_screen = mallocAndCpy(h_screen, screenSize.x * screenSize.y * 3);
    float* d_rect = mallocAndCpy(h_rect, rectSize.x * rectSize.y * 3);
    int* d_ans = checkMalloc<int>(ansSize);

    hipMemcpyToSymbol(HIP_SYMBOL(MATCH_THRESHOLD), &Params::MATCH_THRESHOLD, sizeof(float));



    if (rectSize.x == 32 && rectSize.y == 32 && !forceSlow) {
        int threads = 32 * 32;
        int blockLength = 32;
        int blocks = static_cast<int>(ceil(screen.width() / blockLength) * ceil(screen.height() / blockLength));
        Logger::info("ScreenHelperCUDA::find()", "kernel matchRect32");
        Logger::info("ScreenHelperCUDA::find()", "blocks = " + to_string(blocks) + ", threads = " + to_string(threads));
        matchRect32 <<<blocks, threads >>> (d_screen, screenSize, d_rect, rectSize, d_ans);
    }
    else {
        int threads = 512;
        int blocks = static_cast<int>(ceil(screen.size() / threads));
        Logger::info("ScreenHelperCUDA::find()", "kernel matchRect");
        Logger::info("ScreenHelperCUDA::find()", "blocks = " + to_string(blocks) + ", threads = " + to_string(threads));
        matchRect <<<blocks, threads >>> (d_screen, screenSize, d_rect, rectSize, d_ans);
    }

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        Logger::error("ScreenHelperCUDA::find()", "kernel failed");
    }

    checkCpy(ans, d_ans, ansSize, hipMemcpyDeviceToHost);
    
    hipFree(d_screen);
    hipFree(d_rect);
    hipFree(d_ans);

    for (int i = 0; i < ansSize; i++) {
        if (ans[i]) {
            Logger::info("ScreenHelperCUDA::find()", "!!! found !!!");
            return Point(i % screen.width(), i / screen.width());
        }
    }
    Logger::info("ScreenHelperCUDA::find()", "not found");
    return Point();
}
