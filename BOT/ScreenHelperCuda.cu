#include "hip/hip_runtime.h"
#include "ScreenHelperCUDA.h"
#include "BOT.h"
#include "Params.h"

#include <stdio.h>

using namespace std;

__constant__ float MATCH_THRESHOLD;

__device__ int matches(int num, int area) {
    return (float)num / area >= MATCH_THRESHOLD;
};

__device__ float3 colorAt(float* tab, dim3 tabSize, int x, int y) {
    int idx = y * tabSize.x + x;
    float3 ans;
    ans.x = tab[3 * idx];
    ans.x = tab[3 * idx + 1];
    ans.x = tab[3 * idx + 2];
    return ans;
}

__global__ void matchRect(float* screen, dim3 screenSize, float* rect, dim3 rectSize, int* ans) {
    int idx = blockIdx.x * blockDim.x  + threadIdx.x;
    int hit = 0, cnt = 0;
    int startPointY = idx / screenSize.x;
    int startPointX = idx % screenSize.x;
    int rectArea = rectSize.x * rectSize.y;
    if (idx >= screenSize.x * screenSize.y) {
        return;
    }

    for (int y = startPointY; y < startPointY + rectSize.y && y < screenSize.y; y++) {
        for (int x = startPointX; x < startPointX + rectSize.x && x < screenSize.x; x++) {
            float3 sc = colorAt(screen, screenSize, x, y);
            float3 rc = colorAt(rect, rectSize, x - startPointX, y - startPointY);
            if (sc.x == rc.x && sc.y == rc.y && sc.z == rc.z) {
                hit++;
            }
            cnt++;

            int possibleBest = hit + rectArea - cnt;
            if (!matches(possibleBest, rectArea)) {
                ans[idx] = 0;
                return;
            }
        }
    }
    
    ans[idx] = matches(hit, rectArea);
}

constexpr int MAX_ANS_SIZE = 1920 * 1080 * 42;
int ans[MAX_ANS_SIZE];

Point ScreenHelperCUDA::find(const BmpRect& rect) {
    const Screen& screen = BOT::screen;

    // Choose which GPU to run on, change this on a multi-GPU system.
    if (hipSetDevice(0) != hipSuccess) {
        Logger::error("ScreenHelperCUDA::find()", "hipSetDevice failed!");
    }

    float* h_screen = reinterpret_cast<float*>(screen._screen);
    float* h_rect = reinterpret_cast<float*>(rect._colors);

    dim3 screenSize = dim3(screen.width(), screen.height());
    dim3 rectSize = dim3(rect.width(), rect.height());
    int ansSize = screen.size();

    float* d_screen = mallocAndCpy(h_screen, screenSize.x * screenSize.y * 3);
    float* d_rect = mallocAndCpy(h_rect, rectSize.x * rectSize.y * 3);
    int* d_ans = checkMalloc<int>(ansSize);

    hipMemcpyToSymbol(HIP_SYMBOL(MATCH_THRESHOLD), &Params::MATCH_THRESHOLD, sizeof(float));

    int threads = 512;
    int blocks = static_cast<int>(ceil(screen.size() / threads));
    Logger::info("ScreenHelperCUDA::find()", "blocks = " + to_string(blocks) + ", threads = " + to_string(threads));

    matchRect << <blocks, threads >> > (d_screen, screenSize, d_rect, rectSize, d_ans);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        Logger::error("ScreenHelperCUDA::find()", "kernel failed");
    }

    checkCpy(ans, d_ans, ansSize, hipMemcpyDeviceToHost);
    
    hipFree(d_screen);
    hipFree(d_rect);
    hipFree(d_ans);

    for (int i = 0; i < ansSize; i++) {
        if (ans[i]) {
            Logger::info("ScreenHelperCUDA::find()", "!!! found !!!");
            return Point(i % screen.width(), i / screen.width());
        }
    }
    Logger::info("ScreenHelperCUDA::find()", "not found");
    return Point();
}
